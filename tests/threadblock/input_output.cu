#include "hip/hip_runtime.h"
#include "aso/kernel/graph.h"
#include "aso/threadblock/cuda/input_loader.h"
#include "aso/threadblock/cuda/output_saver.h"
#include "aso/threadblock/graph.h"

#include <fstream>
#include <iostream>

#include <gtest/gtest.h>

#include "common.h"

using namespace aso::threadblock;
using namespace aso::kernel;

__global__ void
    launch_input_output_kernel(DTensor D_In, DTensor D_Out, STensor S_tensor) {
  extern __shared__ char smem_buffer[];
  // save to shared memory and copy back

  int tb_offset_row = 0;
  int tb_offset_column = 0;

  cutlass::MatrixCoord matrix_offset = {tb_offset_row, tb_offset_column};
  int global_offset = 0;
  aso::threadblock::GenericInputLoader loader(smem_buffer,
                                              D_In,
                                              S_tensor,
                                              threadIdx.x,
                                              blockDim.x,
                                              matrix_offset,
                                              global_offset);
  __syncthreads();
  aso::threadblock::GenericOutputSaver saver(smem_buffer,
                                             D_Out,
                                             S_tensor,
                                             threadIdx.x,
                                             blockDim.x,
                                             matrix_offset,
                                             global_offset);
  __syncthreads();
}

TEST(threadblock_tests, input_output) {
  aso::kernel::Graph kgraph;

  // single thread block test
  aso::threadblock::Graph bgraph({1, 1, 1}, {128, 1, 1}, 4);
  aso::kernel::DTensor Input = kgraph.new_input(
      {64, 64}, aso::type::DT_FLOAT16, aso::layout::DmemLayout::DmemRowMajor);
  aso::kernel::DTensor Output = kgraph.new_input(
      {64, 64}, aso::type::DT_FLOAT16, aso::layout::DmemLayout::DmemRowMajor);
  aso::kernel::DTensor Output_Ref = kgraph.new_input(
      {64, 64}, aso::type::DT_FLOAT16, aso::layout::DmemLayout::DmemRowMajor);

  int const num_threads_per_blk = 1024;
  int num_blocks =
      (Input.num_elements() + num_threads_per_blk - 1) / num_threads_per_blk;

  random_fill_device_tensor<cutlass::half_t>
      <<<num_blocks, num_threads_per_blk>>>(Input, Input.num_elements());
  hipMemcpy(Output_Ref.data_ptr,
             Input.data_ptr,
             Input.num_elements() * sizeof(cutlass::half_t),
             hipMemcpyDeviceToDevice);

  aso::threadblock::STensor Input_S =
      bgraph.new_input(Input, {0, -1, -1}, -1, aso::layout::SmemRowMajor);

  int smem_size = 48 * 1024; // 48 KB
  launch_input_output_kernel<<<bgraph.grid_dim, bgraph.block_dim, smem_size>>>(
      Input, Output, Input_S);

  // check Output and Output_Ref
  int h_isEqual = 0;
  int *d_isEqual;

  hipMalloc(&d_isEqual, sizeof(int));
  hipMemcpy(d_isEqual, &h_isEqual, sizeof(int), hipMemcpyHostToDevice);

  // Launch the kernel with the adapted parameters
  checkTensorsEqual<cutlass::half_t><<<num_blocks, num_threads_per_blk>>>(
      Output.data_ptr, Output_Ref.data_ptr, d_isEqual, Output.num_elements());

  // Copy the result back to host
  hipMemcpy(&h_isEqual, d_isEqual, sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "Unequal number of elements: " << h_isEqual << std::endl;
  hipFree(d_isEqual);
}
