#include "hip/hip_runtime.h"
/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "aso/kernel/customized.h"
#include "aso/threadblock/cuda/matmul.h"

namespace aso {
namespace kernel {

__global__ void customized_kernel_function(CustomizedOp::Params const &params) {
  extern __shared__ char smem_buffer[];
  for (int i = 0; i < params.forloop_range; i++) {
    // TODO: prologue for loading data into shared memory
    // start executing operators
    for (int op = 0; op < params.num_operators; op++) {
      if (params.operator_types[op] == aso::type::TB_MATMUL) {
        using ThreadblockShape = cutlass::gemm::GemmShape<64, 64, 32>;
        using WarpShape = cutlass::gemm::GemmShape<32, 32, 32>;
        using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
        int thread_idx = threadIdx.x;
        // Broadcast the warp_id computed by lane 0 to ensure dependent code
        // is compiled as warp-uniform.
        int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
        int lane_idx = threadIdx.x % 32;

        aso::threadblock::matmul::MatmulExecutor<ThreadblockShape,
                                                 WarpShape,
                                                 InstructionShape,
                                                 cutlass::half_t,
                                                 cutlass::layout::RowMajor,
                                                 cutlass::layout::ColumnMajor>
            executor(thread_idx, warp_idx, lane_idx);
        executor.compute_kernel();
      }
    }
  }
}

void CustomizedOp::run() {
  int smem_size = 48 * 1024 * 1024;
  Params params;
  customized_kernel_function<<<plan.grid_dim, plan.block_dim, smem_size>>>(
      params);
}

bool CustomizedOp::profile(ProfileResult &result) {}

} // namespace kernel
} // namespace aso
