/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "aso/kernel/device_memory_manager.h"
#include "aso/utils/cuda_helper.h"

namespace aso {
namespace kernel {

DeviceMemoryManager *DeviceMemoryManager::singleton = nullptr;

DeviceMemoryManager::DeviceMemoryManager() {
  // preallocate 10 GB of device memory
  total_size = (size_t)10 * 1024 * 1024 * 1024;
  offset = 0;
  checkCUDA(hipMalloc(&base_ptr, total_size));
  checkCUDA(hipblasCreate(&blas));
  checkCUDA(hipblasSetMathMode(blas, HIPBLAS_TENSOR_OP_MATH));
}

DeviceMemoryManager::~DeviceMemoryManager() {
  checkCUDA(hipFree(base_ptr));
  checkCUDA(hipblasDestroy(blas));
}

void *DeviceMemoryManager::allocate(size_t size_in_bytes) {
  void *ret_ptr = base_ptr + offset;
  offset += size_in_bytes;
  // Assert that we haven't used more than what we pre-allocated
  assert(offset <= total_size);
  allocated_tensors.push_back(std::make_pair(ret_ptr, size_in_bytes));
  return ret_ptr;
}

void DeviceMemoryManager::free(void *ptr) {
  // Currently assume that tensors are freed in the reverse order
  // so ptr must be the last tensor we have created
  assert(allocated_tensors.size() > 0);
  assert(allocated_tensors.back().first == ptr);
  offset -= allocated_tensors.back().second;
  allocated_tensors.pop_back();
}

DeviceMemoryManager *DeviceMemoryManager::get_instance() {
  if (singleton == nullptr) {
    singleton = new DeviceMemoryManager();
  }
  return singleton;
}

} // namespace kernel
} // namespace aso
