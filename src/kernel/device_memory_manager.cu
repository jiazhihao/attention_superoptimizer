/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "aso/kernel/device_memory_manager.h"
#include "aso/utils/cuda_helper.h"

namespace aso {
namespace kernel {

DeviceMemoryManager *DeviceMemoryManager::singleton = nullptr;

DeviceMemoryManager::DeviceMemoryManager() {
  // preallocate 10 GB of device memory
  total_size = (size_t)10 * 1024 * 1024 * 1024;
  offset = 0;
  checkCUDA(hipMalloc(&base_ptr, total_size));
  checkCUDA(hipblasCreate(&blas));
  checkCUDA(hipblasSetMathMode(blas, HIPBLAS_TENSOR_OP_MATH));
}

DeviceMemoryManager::~DeviceMemoryManager() {
  checkCUDA(hipFree(base_ptr));
  checkCUDA(hipblasDestroy(blas));
}

bool DeviceMemoryManager::allocate(DTensor &tensor, bool allocate_fingerprint) {
  void *ret_ptr = base_ptr + offset;
  offset += tensor.data_size();
  tensor.data_ptr = ret_ptr;
  allocated_tensors.push_back(std::make_pair(ret_ptr, tensor.data_size()));

  if (allocate_fingerprint) {
    ret_ptr = base_ptr + offset;
    offset += tensor.fingerprint_size();
    tensor.fp_ptr = (aso::type::FPType *)ret_ptr;
    allocated_tensors.push_back(
        std::make_pair(ret_ptr, tensor.fingerprint_size()));
  }
  // Assert that we haven't used more than what we pre-allocated
  assert(offset <= total_size);

  return true;
}

bool DeviceMemoryManager::free(DTensor &tensor) {
  // Currently assume that tensors are freed in the reverse order
  // so ptr must be the last tensor we have created
  if (tensor.fp_ptr != nullptr) {
    assert(allocated_tensors.size() > 0);
    assert(allocated_tensors.back().first == tensor.fp_ptr);
    offset -= allocated_tensors.back().second;
    allocated_tensors.pop_back();
  }
  assert(allocated_tensors.size() > 0);
  assert(allocated_tensors.back().first == tensor.data_ptr);
  offset -= allocated_tensors.back().second;
  allocated_tensors.pop_back();
  return true;
}

DeviceMemoryManager *DeviceMemoryManager::get_instance() {
  if (singleton == nullptr) {
    singleton = new DeviceMemoryManager();
  }
  return singleton;
}

} // namespace kernel
} // namespace aso
