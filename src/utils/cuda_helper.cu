/* Copyright 2023-2024 CMU
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "mirage/utils/cuda_helper.h"

namespace mirage {
namespace utils {

hipDataType to_cuda_datatype(mirage::type::DataType type) {
  switch (type) {
    case mirage::type::DT_FLOAT16:
      return HIP_R_16F;
    case mirage::type::DT_FLOAT32:
      return HIP_R_32F;
    case mirage::type::DT_DOUBLE:
      return HIP_R_64F;
    default:
      assert(false && "Unspoorted cuda data type");
  }
  return HIP_R_16F;
}

size_t get_max_shared_mem() {
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProps;
  hipGetDeviceProperties(&deviceProps, device);
  return deviceProps.sharedMemPerBlock;
}

} // namespace utils
} // namespace mirage
